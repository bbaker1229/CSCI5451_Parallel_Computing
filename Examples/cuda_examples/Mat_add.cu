
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BSIZE  16
#define NN 4000;

__global__ void MatAdd(int N, float *A, float *B, float *C){ 
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   if (i < N && j < N)
       C[i*N+j] = A[i*N+j] + B[i*N+j]; 
} 

 void err_exit(char *message);
 float mat_add_check(int n, float *x, float *y, float *z)  {
 float s=0.0, t = 0.0, td = 0.0;
 for (int i=0; i<n; i++) {
       s  = y[i]+x[i]-z[i]; 
       t += s*s ;
       td += (x[i]*x[i]+y[i]*y[i]);
 }    
//-------------------- matrices are both zero
 if (td == 0.0) return(-1);
    else
//-------------------- normal return
   return(sqrt(s/td));
} 

int main() {
float *Ad, *Bd, *Cd; 
float  *A,  *B,  *C; 
int N, i, j; 
size_t MatSize;
float s;
//-------------------- set dimension N
 N = NN;

 char LineG[] = "Error allocating GPU  memory";
 char LineH[] = "Error allocating Host memory";

  
 MatSize = N*N*sizeof(float);
//-------------------- allocate on cpu
 A = (float *)malloc(MatSize);        
 B = (float *)malloc(MatSize);        
 C = (float *)malloc(MatSize);    
 if ((A==NULL) | (B==NULL) | (C==NULL) ) 
          err_exit(LineH);
//-------------------- allocate on GPU
 if (hipMalloc((void **) &Ad, MatSize) != hipSuccess) 
       err_exit(LineG);
 if (hipMalloc((void **) &Bd, MatSize) != hipSuccess) 
       err_exit(LineG);
 if (hipMalloc((void **) &Cd, MatSize) != hipSuccess) 
       err_exit(LineG);
//-------------------- fill arrays A,B

 for (i=0; i<N; i++) 
    for (j=0; j<N; j++) {
      A[i*N+j] = (float) rand() / (float) rand();
      B[i*N+j] = (float) rand() / (float) rand();
} 
//
//-------------------- copy matrices A,B+ to GPU memory
hipMemcpy(Ad, A, MatSize, hipMemcpyHostToDevice);
hipMemcpy(Bd, B, MatSize, hipMemcpyHostToDevice);
//-------------------- Kernel invocation
   dim3 dimBlock(BSIZE, 256/BSIZE);
   dim3 dimGrid((N + dimBlock.x-1) / dimBlock.x,
                (N + dimBlock.y-1) / dimBlock.y);
   MatAdd<<<dimGrid, dimBlock>>>(N, Ad, Bd, Cd);
//-------------------- see if things did execute 
 hipError_t error = hipGetLastError();
 if (error) {
     printf("CUDA error: %s \n",hipGetErrorString(error));
     exit(1);
 }
//-------------------- Transfer result from GPU to CPU
hipMemcpy(C, Cd, MatSize, hipMemcpyDeviceToHost);
//-------------------- check whether addition was correct
s =  mat_add_check(N*N,A,B,C);
 
printf(" Mat dim = %d -- err= %10.6e\n",N,s); 
//-------------------- Free Host arrays
 free(A); 
 free(B);
 free(C);
//-------------------- Free GPU memory
 hipFree(Ad);
 hipFree(Bd);
 hipFree(Cd);	
}

//-------------------- Prints error error Msg and exits 
void err_exit(char *errMsg) {
	printf("%s\n", errMsg);
	exit(1);
}
