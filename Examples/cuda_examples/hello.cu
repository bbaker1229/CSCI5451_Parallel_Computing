
#include <hip/hip_runtime.h>
#include <stdio.h>
  __global__ void helloFromGPU(){
    printf("Hello World-Thread: %d\n",threadIdx.x);                                                      
  }

  int main(void) {
    helloFromGPU<<<1,16>>>();
    hipDeviceSynchronize();
    return(0);
  }
