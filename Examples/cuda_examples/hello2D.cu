
#include <hip/hip_runtime.h>
#include <stdio.h>
  __global__ void helloFromGPU2(){
  int i = threadIdx.x;
  int j = threadIdx.y;
  printf("Hello World-Thread: %d, %d \n",i,j);                                                      
  }

  int main(void) {
  dim3 ThisBlock(4,4);
    helloFromGPU2<<<1,ThisBlock>>>();
    hipDeviceSynchronize();
    return(0);
  }
