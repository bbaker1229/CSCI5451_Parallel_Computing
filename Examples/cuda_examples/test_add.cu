#include <stdio.h>
#include <hip/hip_runtime.h>
	   
   __global__ void vec_add(float *a, float *b)   {
//  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
      a[i] = a[i]+b[i];
    }
/*-------------------- main will execute on host */

int main(void){
/* -------------------- host & device arrays*/
  float *y_h, *y_d,  *x_h, *x_d;  
/* -------------------- size of arrays */
  const int N = 500;  
  size_t size = N * sizeof(float);
/* -------------------- Allocate array on host */
  y_h = (float *)malloc(size);        
  x_h = (float *)malloc(size);        
/* -------------------- Allocate array on device */
  hipMalloc((void **) &y_d, size);   
  hipMalloc((void **) &x_d, size);   
/* -------------------- Initialize host array & 
                        copy it to device */
  for (int i=0; i<N; i++) {
      y_h[i] = (float) i;
      x_h[i] = (float) (N-i);
  }     
  hipMemcpy(y_d,y_h,size,hipMemcpyHostToDevice);
  hipMemcpy(x_d,x_h,size,hipMemcpyHostToDevice);
/* -------------------- invoke kernel on device */
//   int block_size = 4;
//   int n_blocks = (N+block_size-1)/block_size; 
//   vec_add <<< n_blocks, block_size >>> (y_d, N);
    vec_add <<< 1, N >>> (y_d, x_d);    
/*-------------------- retrieve result from device*/
   hipMemcpy(y_h,y_d,size,hipMemcpyDeviceToHost);
/* --------------------/ Print 20 first results / */
   for (int i=0; i<20; i++) 
       printf("%d %f\n", i, y_h[i]);
/* -------------------- free memory */ 
   free(y_h); hipFree(y_d);
 }

