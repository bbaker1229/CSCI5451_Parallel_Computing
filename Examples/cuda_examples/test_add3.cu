#define NITER 1
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
	   
/*-------------------- CUDA kernel. does an add  */
   __global__ void vec_add3(float *a, float *b, int n) {
  int block_id = gridDim.x * blockIdx.y + blockIdx.x ; 
  int i_t = blockDim.x * block_id + threadIdx.x;
  if (i_t<n) 	       
      a[i_t] += b[i_t] ; 
 }    
/*-------------------- main will execute on host */

int main(void){
/* -------------------- host & device arrays */
  float *y_h, *y_d,  *x_h, *x_d;  
  hipError_t err;
  float ferr;
/* -------------------- size of arrays */
  const int N = 140000;
  int block_size ;
  int num_blocks_x, num_blocks_y;
  int i,len = 128*1024;
  size_t size = N * sizeof(float);
/* -------------------- Allocate array on host */
  y_h = (float *)malloc(size);        
  x_h = (float *)malloc(size);        
/* -------------------- Allocate array on device */
  hipMalloc((void **) &y_d, size);   
  hipMalloc((void **) &x_d, size);   
/* -------------------- Initialize host array & 
                        copy it to device */  
  for (int i=0; i<len; i++) {
      y_h[i] = (float) i / (float)len;
      x_h[i] = (float) (len-i)/(float)len;
  }     
  hipMemcpy(y_d,y_h,size,hipMemcpyHostToDevice);
  hipMemcpy(x_d,x_h,size,hipMemcpyHostToDevice);
/* -------------------- invoke kernel        */
 block_size = 1024;
 num_blocks_x = 16;	
 num_blocks_y = 8;
 dim3 grid_size(num_blocks_x, num_blocks_y, 1);
 vec_add3 <<< grid_size,block_size >>> (x_d, y_d,len);
//-------------------- get last error message
    err = hipGetLastError() ;
    if (err != hipSuccess) 
    printf(" error:  %s\n",hipGetErrorString(err));
/*-------------------- get result from device */
   hipMemcpy(x_h,x_d,size,hipMemcpyDeviceToHost);
/* --------------------/ Print 20 first results / */
   ferr = 0.0;
   for (i=0; i<len; i++) 
      ferr += (1.0 - y_h[i])*(1.0 - x_h[i]);
   printf(" Done len = %d  \n Abs. Error is : %f\n", len, ferr);

/* -------------------- free memory */ 
   free(x_h); hipFree(x_d);
   free(y_h); hipFree(y_d);
 }

