
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NITER 100

__global__ void saxpy_par(unsigned long int n, float a, float *x, float *y) {
  //
  unsigned long int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
    y[i] += a*x[i];
}

float saxpy_check(unsigned long int n, float a, float *x, float *y, float *z) {
  // a, x, y == original data for saxpy
  // z = result found -- with which to compare.
  float s=0.0, t=0.0;
  for (unsigned long int i=0; i<n; i++) {
    y[i] += a * x[i];
    s += (y[i] - z[i]) * (y[i] - z[i]);
    t += z[i] * z[i];
  }
  if (t == 0.0) return (-1);
  else
    return(sqrt(s/t));
}

int main(int argc, char *argv[]) {
  unsigned long int n = 8 * 1024 * 1024;
  unsigned long int i;//, maxval;
  float a, value, valresult;
  float *x, *y, *z, *xg, *yg;//, *zg, *xtmp, *ytmp, *ztmp;
  a = 1.0;
  a = a / (float) NITER;
  x = (float*) malloc(n*sizeof(float));
  y = (float*) malloc(n*sizeof(float));
  z = (float*) malloc(n*sizeof(float));
  for(i=0; i < n; i++) {
    value = (float)rand() / (float)RAND_MAX;
    x[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    y[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    z[i] = value;
  }
  //printf("Entering loop:\n");
  for (unsigned long int vecLen = 2048; vecLen <= n; vecLen *= 2) {
    //printf("vecLen = %lu\n", vecLen);
    /*xtmp = (float*) malloc(vecLen*sizeof(float));
    ytmp = (float*) malloc(vecLen*sizeof(float));
    ztmp = (float*) malloc(vecLen*sizeof(float));
    if(2*vecLen > n)
      maxval = n;
    else
      maxval = 2*vecLen;
    for(i=vecLen; i <= maxval; i++) {
      xtmp[i-vecLen] = x[i];
      ytmp[i-vecLen] = y[i];
      ztmp[i-vecLen] = z[i];
    }*/
    //printf("Copied data\n");
    hipMalloc(&xg, vecLen*sizeof(float));
    hipMalloc(&yg, vecLen*sizeof(float));
    //cudaMalloc(&zg, vecLen*sizeof(float));
    hipMemcpy(xg, x, vecLen*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(yg, y, vecLen*sizeof(float), hipMemcpyHostToDevice);
    //cudaMemcpy(zg, z, vecLen*sizeof(float), cudaMemcpyHostToDevice);
    dim3 dimGrid(1024 * 8);
    dim3 dimBlock(1024);
    for(int iter=0; iter<NITER; iter++){
      saxpy_par<<<dimGrid, dimBlock>>>(vecLen, a, xg, yg);
      hipMemcpy(z, yg, vecLen*sizeof(float), hipMemcpyDeviceToHost);
      valresult = saxpy_check(vecLen, a, x, y, z);
    }
    printf("** vecLen = %7.0lu, Mflops = xx.dd  err = %2.2e\n", vecLen, valresult);
  }
  //printf("working\n");
  return(0);
}
