
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>

#define NITER 100 // The number of iterations to use kernel function.

__global__ void saxpy_par(unsigned long int n, float a, float *x, float *y) {
  // Cuda kernel function for saxpy.
  // Result is stored back into y.
  unsigned long int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
    y[i] += a*x[i];
}

float saxpy_check(unsigned long int n, float a, float *x, float *y, float *z) {
  // a, x, y == original data for saxpy
  // z = result found -- with which to compare.
  float s=0.0, t=0.0;
  for (unsigned long int i=0; i<n; i++) {
    y[i] += a * x[i];
    s += (y[i] - z[i]) * (y[i] - z[i]);
    t += z[i] * z[i];
  }
  if (t == 0.0) return (-1);
  else
    return(sqrt(s/t));
}

double wctime() {
  // calculate wall time.
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main(int argc, char *argv[]) {
  unsigned long int n = 8 * 1024 * 1024;
  unsigned long int i;
  float a, value, valresult, nops;
  float *x, *y, *z, *xg, *yg;
  double t1[NITER];
  double avgt;
  a = 1.0;
  a = a / (float) NITER;
  // Allocate memory for vectors.
  x = (float*) malloc(n*sizeof(float));
  y = (float*) malloc(n*sizeof(float));
  z = (float*) malloc(n*sizeof(float));
  // Fill vectors will random data.
  for(i=0; i < n; i++) {
    value = (float)rand() / (float)RAND_MAX;
    x[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    y[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    z[i] = value;
  }
  // Begin main for loop for testing vector length.
  for (unsigned long int vecLen = 2048; vecLen <= n; vecLen *= 2) {
    // Allocate memory on GPU for x and y.
    hipMalloc(&xg, vecLen*sizeof(float));
    hipMalloc(&yg, vecLen*sizeof(float));
    // Copy vectors given a size of vecLen to the GPU.
    hipMemcpy(xg, x, vecLen*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(yg, y, vecLen*sizeof(float), hipMemcpyHostToDevice);
    // Define the cuda grid and block size.
    dim3 dimGrid(1024 * 8);
    dim3 dimBlock(1024);
    // Run saxpy kernel function NITER times on the GPU and collect time.
    for(int iter=0; iter<NITER; iter++){
      t1[iter] = wctime();
      saxpy_par<<<dimGrid, dimBlock>>>(vecLen, a, xg, yg);
      t1[iter] = (wctime() - t1[iter])*1.e+06;
    }
    // Copy result from GPU to CPU.
    hipMemcpy(z, yg, vecLen*sizeof(float), hipMemcpyDeviceToHost);
    // Calculate the error with check on CPU.
    valresult = saxpy_check(vecLen, a * NITER, x, y, z);
    // Find the average time for each NITER.
    for(i=0; i<NITER; i++) {
      avgt += t1[i];
    }
    avgt /= (float) NITER;
    // Find the number of floating point ops.
    nops = (float) vecLen * 2;
    // Print results.
    printf("** vecLen = %7.0lu, Mflops = %10.2lf  err = %2.2e\n", vecLen, nops/avgt, valresult);
    // Free memory on GPU.  
    hipFree(xg);
    hipFree(yg);
  }
  // Free memory on CPU.  
  free(x);
  free(y);
  free(z);
  return(0);
}
