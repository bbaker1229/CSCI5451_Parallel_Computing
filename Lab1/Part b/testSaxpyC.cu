
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>

#define NITER 100

__global__ void saxpy_par(unsigned long int n, float a, float *x, float *y) {
  unsigned long int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
    y[i] += a*x[i];
}

float saxpy_check(unsigned long int n, float a, float *x, float *y, float *z) {
  // a, x, y == original data for saxpy
  // z = result found -- with which to compare.
  float s=0.0, t=0.0;
  for (unsigned long int i=0; i<n; i++) {
    y[i] += a * x[i];
    s += (y[i] - z[i]) * (y[i] - z[i]);
    t += z[i] * z[i];
  }
  if (t == 0.0) return (-1);
  else
    return(sqrt(s/t));
}

double wctime()
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main(int argc, char *argv[]) {
  unsigned long int n = 8 * 1024 * 1024;
  unsigned long int i;
  float a, value, valresult, nops;
  float *x, *y, *z, *xg, *yg;
  double t1[NITER];
  double avgt;
  a = 1.0;
  //a = a / (float) NITER;
  x = (float*) malloc(n*sizeof(float));
  y = (float*) malloc(n*sizeof(float));
  z = (float*) malloc(n*sizeof(float));
  for(i=0; i < n; i++) {
    value = (float)rand() / (float)RAND_MAX;
    x[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    y[i] = value;
    value = (float)rand() / (float)RAND_MAX;
    z[i] = value;
  }
  for (unsigned long int vecLen = 2048; vecLen <= n; vecLen *= 2) {
    hipMalloc(&xg, vecLen*sizeof(float));
    hipMalloc(&yg, vecLen*sizeof(float));
    //cudaMemcpy(xg, x, vecLen*sizeof(float), cudaMemcpyHostToDevice);
    //cudaMemcpy(yg, y, vecLen*sizeof(float), cudaMemcpyHostToDevice);
    dim3 dimGrid(1024 * 8);
    dim3 dimBlock(1024);
    //t1 = wctime();
    for(int iter=0; iter<NITER; iter++){
      t1[iter] = wctime();
      hipMemcpy(xg, x, vecLen*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(yg, y, vecLen*sizeof(float), hipMemcpyHostToDevice);
      saxpy_par<<<dimGrid, dimBlock>>>(vecLen, a, xg, yg);
      hipMemcpy(z, yg, vecLen*sizeof(float), hipMemcpyDeviceToHost);
      t1[iter] = (wctime() - t1[iter])*1.e+06;
      //valresult = saxpy_check(vecLen, a, x, y, z);
    }
    //t1 = (wctime() - t1)*1.e+06;
    for(i=0; i<NITER; i++)
      avgt += t1[i];
    avgt /= (float) NITER;
    valresult = saxpy_check(vecLen, a, x, y, z);
    nops = (float) vecLen * 2;
    printf("** vecLen = %7.0lu, Mflops = %10.2lf  err = %2.2e\n", vecLen, nops/avgt, valresult);
    hipFree(xg);
    hipFree(yg);
  }
  free(x);
  free(y);
  free(z);
  return(0);
}
